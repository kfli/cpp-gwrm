#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <time.h>
#include <sys/time.h>
typedef double df;
#define USECPSEC 1000000ULL
#define BSX 1<<5
#define BSY 1<<5
#define N 100
#define M 100

const bool sync = true;
const bool nosync = false;
unsigned long long dtime_usec(unsigned long long start, bool use_sync = nosync){
  if (use_sync == sync) hipDeviceSynchronize();
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

int divUp(int a, int b) {return (a + b - 1) / b;}

float cpu_sum(int n, int m, df *a, df *b, df *c) {
   df q, r;
   #pragma omp parallel for collapse(2)
   for (int x = 0; x < n; x++) {
      for (int y = 0; y < m; y++) {
         q = 0.0f;
         for (int i = 0; i <= x; i++) {
            r = 0.0f;
            for (int j = 0; j <= y; j++) {
               r += a[i * n + j] * b[(x - i) * n + y - j];
            }
            for (int j = 1; j < m - y; j++) {
               r += a[i * n + j] * b[(x - i) * n + y + j] 
                    + a[i * n + y + j] * b[(x - i) * n + j];
            }
            q += r;
         }
         for (int i = 1; i < n-x; i++) {
            r = 0.0f;
            for (int j = 0; j <= y; j++) {
               r += a[i * n + j] * b[(x + i) * n + y - j]
                    + a[(x + i) * n + j] * b[ i * n + y - j];
            }
            for (int j = 1; j < m - y; j++) {
               r += a[i * n + j] * b[(x + i) * n + y + j] 
                    + a[(x + i) * n + y + j] * b[(x + i) * n + j]
                 
                    +a[(x + i) * n + j] * b[i * n + y + j] 
                    + a[(x + i) * n + y + j] * b[i * n + j];
            }
            q += r;
         }
      c[x * N + y] = 0.25f*q;
      }
   }
   return 0;
}

const int P2  = 5;
const int TPB = 1<<P2;
const unsigned row_mask = ~((0xFFFFFFFFU>>P2)<<P2);
__global__ void chebyprod_imp(int n, int m, df *a, df *b, df *c){
   __shared__ df sdata_x[TPB], sdata_y[TPB];
   int x = blockIdx.x;
   int y = blockIdx.y;
   int row_width_x = (((x)>(n-x))?(x):(n-x))+1;
   int row_width_y = (((y)>(m-y))?(y):(m-y))+1;
   int strides_x = (row_width_x>>P2) + ((row_width_x&row_mask)?1:0);
   int strides_y = (row_width_y>>P2) + ((row_width_y&row_mask)?1:0);
   int i = threadIdx.x;
   df tmp_a, r;
   df sum = 0.0f;
   for (int s=0; s < strides_x; s++) { // block-stride x loop
      if (i <= x) {
         int j = threadIdx.y;
         r = 0.0f;
         for (int u=0; u < strides_y; u++) { // block-stride y loop
            if (i < n && j < m) {tmp_a = a[i * n + j];}
            if (j <= y) {r += tmp_a * b[(x - i) * n + y - j];}
            if ((j > 0) && (j < (m-y))) {r += tmp_a * b[(x - i) * n + y + j] 
                                             + a[i * n + y + j] * b[(x - i) * n + j];}
            j += TPB;
         }
         sdata_y[threadIdx.y] = r;
         for (int u = TPB>>1; u > 0; u>>=1) { // sweep reduction in y
            __syncthreads();
            if (threadIdx.y < u) {
               sdata_y[threadIdx.y] += sdata_y[threadIdx.y + u];
            }
         }
         if (!threadIdx.y) sum += sdata_y[0];
      }
      if ((i > 0) && (i < (n-x))) {
         int j = threadIdx.y;
         r = 0.0f;
         for (int u=0; u < strides_y; u++) { // block-stride y loop
            if (i < n && j < m) {tmp_a = a[i * n + j];}     
            if (j <= y) {r += tmp_a * b[(x + i) * n + y - j]
                              + a[(x + i) * n + j] * b[ i * n + y - j];}
            if ((j > 0) && (j < (m-y))) {r += tmp_a * b[(x + i) * n + y + j] 
                                              + a[(x + i) * n + y + j] * b[(x + i) * n + j]
                                              + a[(x + i) * n + j] * b[i * n + y + j] 
                                              + a[(x + i) * n + y + j] * b[i * n + j];}
            j += TPB;
         }
         sdata_y[threadIdx.y] = r;
         for (int u = TPB>>1; u > 0; u>>=1) { // sweep reduction in y
            __syncthreads();
            if (threadIdx.y < u) {
               sdata_y[threadIdx.y] += sdata_y[threadIdx.y + u];
            }
         }
         if (!threadIdx.y) sum += sdata_y[0];
      }
      i += TPB; 
   }
   sdata_x[threadIdx.x] = sum;
   for (int s = TPB>>1; s > 0; s>>=1) { // sweep reduction in x
         __syncthreads();
         if (threadIdx.x < s) {
            sdata_x[threadIdx.x] += sdata_x[threadIdx.x + s];
         }
   }
   if (!threadIdx.x && !threadIdx.y) c[x * n + y] = 0.25f*sdata_x[0];
}

__global__ void chebyprod(int n, int m, df *a, df *b, df *c){
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   df q, r;
   if (x < n && y < m) {
      q = 0.0f;
      for (int i = 0; i <= x; i++) {
         r = 0.0f;
         for (int j = 0; j <= y; j++) {
            r += a[i * n + j] * b[(x - i) * n + y - j];
         }
         for (int j = 1; j < m - y; j++) {
            r += a[i * n + j] * b[(x - i) * n + y + j] 
                 + a[i * n + y + j] * b[(x - i) * n + j];
         }
         q += r;
      }
      for (int i = 1; i < n-x; i++) {
         r = 0.0f;
         for (int j = 0; j <= y; j++) {
            r += a[i * n + j] * b[(x + i) * n + y - j]
                 + a[(x + i) * n + j] * b[ i * n + y - j];
         }
         for (int j = 1; j < m - y; j++) {
            r += a[i * n + j] * b[(x + i) * n + y + j] 
                 + a[(x + i) * n + y + j] * b[(x + i) * n + j]
                 
                 +a[(x + i) * n + j] * b[i * n + y + j] 
                 + a[(x + i) * n + y + j] * b[i * n + j];
         }
         q += r;
      }
   c[x * N + y] = 0.25f*q;
   }
}

int main(void){
  int size = N*M*sizeof(df);
  df *a, *b, *c, *cc, *ci, *d_a, *d_b, *d_c, *d_ci;
  a  = (df*)malloc(size);
  b  = (df*)malloc(size);
  c  = (df*)malloc(size);
  cc = (df*)malloc(size);
  ci = (df*)malloc(size);

  hipMalloc(&d_a, size); 
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);
  hipMalloc(&d_ci, size);
  #pragma omp parallel for collapse (2)
  for (int i = 0; i < N; i++) {
     for (int j = 0; j < M; j++) {
        a[i * M + j] = 0.1f;
        b[i * M + j] = 0.2f;
     }
  }

  unsigned long long  dt = dtime_usec(0);
  // Perform chebyprod on N elements
  cpu_sum(N, M, a, b, cc);
  dt = dtime_usec(dt,sync);
  printf("Time taken 2D CPU: %fs\n", dt/(float)USECPSEC);
  df dtc = dt/(float)USECPSEC;

  std::cout << "Vector cc: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << cc[k] << " ";
  std::cout <<"]\n";

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  dim3 dimBlock(BSX, BSY);
  dim3 dimGrid(divUp(N, BSX), divUp(M, BSY)); 
  
  //std::cout << "dimBlock: " << dimBlock << "\n dimGrid: " << dimGrid << "\n";
  dt = dtime_usec(0);
  // Perform chebyprod on N elements
  chebyprod<<< dimBlock, dimGrid >>>(N, M, d_a, d_b, d_c);
  dt = dtime_usec(dt,sync);
  printf("Time taken 2D monolithic kernel: %fs\n", dt/(float)USECPSEC);
  printf("Speedup: %fs\n", dtc/(dt/(float)USECPSEC));
  
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  
  std::cout << "Vector c: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << c[k] << " ";
  std::cout <<"]\n";
  
  dt = dtime_usec(0);
  // Perform chebyprod on N elements
  chebyprod_imp<<< dimBlock, dimGrid >>>(N, M, d_a, d_b, d_ci);
  dt = dtime_usec(dt,sync);
  printf("Time taken 2D stride kernel: %fs\n", dt/(float)USECPSEC);

  hipMemcpy(ci, d_ci, size, hipMemcpyDeviceToHost);
  
  std::cout << "Vector ci: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << ci[k] << " ";
  std::cout <<"]\n";
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c); 
  hipFree(d_ci);
  free(a);
  free(b);
  free(c);
  free(cc);
  free(ci);
}
