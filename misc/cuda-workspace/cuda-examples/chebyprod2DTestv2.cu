#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
#include <time.h>
#include <sys/time.h>
typedef double df;
#define USECPSEC 1000000ULL
#define BS 1<<5
#define N 100
#define M 100

const bool sync = true;
const bool nosync = false;
unsigned long long dtime_usec(unsigned long long start, bool use_sync = nosync){
  if (use_sync == sync) hipDeviceSynchronize();
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

float cpu_sum(int n, int m, df *a, df *b, df *c) {
   df q, r;
   #pragma omp parallel for collapse(2)
   for (int x = 0; x < n; x++) {
      for (int y = 0; y < m; y++) {
         q = 0.0f;
         for (int i = 0; i <= x; i++) {
            r = 0.0f;
            for (int j = 0; j <= y; j++) {
               r += a[i * n + j] * b[(x - i) * n + y - j];
            }
            for (int j = 1; j < m - y; j++) {
               r += a[i * n + j] * b[(x - i) * n + y + j] 
                    + a[i * n + y + j] * b[(x - i) * n + j];
            }
            q += r;
         }
         for (int i = 1; i < n-x; i++) {
            r = 0.0f;
            for (int j = 0; j <= y; j++) {
               r += a[i * n + j] * b[(x + i) * n + y - j]
                    + a[(x + i) * n + j] * b[ i * n + y - j];
            }
            for (int j = 1; j < m - y; j++) {
               r += a[i * n + j] * b[(x + i) * n + y + j] 
                    + a[(x + i) * n + y + j] * b[(x + i) * n + j]
                 
                    +a[(x + i) * n + j] * b[i * n + y + j] 
                    + a[(x + i) * n + y + j] * b[i * n + j];
            }
            q += r;
         }
      c[x * N + y] = 0.25f*q;
      }
   }
   return 0;
}

const int P2  = 5;
const int TPB = 1<<P2;
const unsigned row_mask = ~((0xFFFFFFFFU>>P2)<<P2);
__global__ void chebyprod_imp(int n, int m, df *a, df *b, df *c){
   __shared__ df sdata[TPB];
   int x = blockIdx.x;
   int row_width_x = (((x)>(n-x))?(x):(n-x))+1;
   int row_width_y = (((y)>(m-y))?(y):(m-y))+1;
   int strides_x = (row_width_x>>P2) + ((row_width_x&row_mask)?1:0);
   int strides_y = (row_width_y>>P2) + ((row_width_y&row_mask)?1:0);
   int i = threadIdx.x;
   df tmp_a, r;
   df sum = 0.0f;
   for (int s=0; s < strides_x; s++){ // block-stride x loop
      if (i < n && j < m) {tmp_a = a[i * n + j];}
      if (i <= x) {
         r = 0.0f;
            if (j <= y) {r += tmp_a * b[(x - i) * n + y - j];}
            if ((j > 0) && (j < (m-y))) {r += tmp_a * b[(x - i) * n + y + j] 
                                              + a[i * n + y + j] * b[(x - i) * n + j];}
            sum += r;
         }
      }
      if ((i > 0) && (i < (n-x))) {	
         r = 0.0f;
            if (j <= y) {r += tmp_a * b[(x + i) * n + y - j]
                              + a[(x + i) * n + j] * b[ i * n + y - j];}
            if ((j > 0) && (j < (m-y))) {r += tmp_a * b[(x + i) * n + y + j] 
                                              + a[(x + i) * n + y + j] * b[(x + i) * n + j]
                                              + a[(x + i) * n + j] * b[i * n + y + j] 
                                              + a[(x + i) * n + y + j] * b[i * n + j];}
            sum += r;         }
      }
      i += TPB;
   }
   sdata[threadIdx.x * n] = sum;
   for (int s = TPB>>1; s > 0; s>>=1) { // sweep reduction in x
      __syncthreads();
      if (threadIdx.x < s) {
         sdata[threadIdx.x] += sdata[threadIdx.x  + s];
      }
   }
   if (!threadIdx.x) c[x * n ] = 0.25f*sdata[0];
}

int main(void){
  int size = N*M*sizeof(df);
  df *a, *b, *c, *cc, *d_a, *d_b, *d_c;
  a  = (df*)malloc(size);
  b  = (df*)malloc(size);
  c  = (df*)malloc(size);
  cc = (df*)malloc(size);

  hipMalloc(&d_a, size); 
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);
  #pragma omp parallel for collapse (2)
  for (int i = 0; i < N; i++) {
     for (int j = 0; j < M; j++) {
        a[i * M + j] = 0.1f;
        b[i * M + j] = 0.2f;
     }
  }

  unsigned long long  dt = dtime_usec(0);
  // Perform chebyprod on N elements
  cpu_sum(N, M, a, b, cc);
  dt = dtime_usec(dt,sync);
  printf("Time taken 2D CPU: %fs\n", dt/(float)USECPSEC);
  df dtc = dt/(float)USECPSEC;

  std::cout << "Vector cc: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << cc[k] << " ";
  std::cout <<"]\n";

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  const int GS = (N*M + BS -1) / BS;
  
  dt = dtime_usec(0);
  // Perform chebyprod on N elements
  chebyprod_imp<<< GS, BS >>>(N, M, d_a, d_b, d_c);
  dt = dtime_usec(dt,sync);
  printf("Time taken 2D stride kernel: %fs\n", dt/(float)USECPSEC);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  
  std::cout << "Vector c: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << c[k] << " ";
  std::cout <<"]\n";
 
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c); 
  free(a);
  free(b);
  free(c);
  free(cc);
}
