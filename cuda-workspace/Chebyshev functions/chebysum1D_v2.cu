#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>
//typedef double mt;

/*
__global__ void chebyprod(int n, float *a, float *b, float *c){
   int i = blockIdx.x *blockDim.x + threadIdx.x;
   float sum;
   if (i < n) {
      sum = 0.f;
      for (int j = 0; j<=i; j++){
         sum += a[j]*b[j-i];
      }
      for (int j = 1; j < n-i; j++){
         sum += a[j]*b[j+i]+a[j+i]*b[j];
      }
      c[i] = 0.5f*sum;
   }  
}
*/

template <int blockSize>
__global__ void child_sum(int n, int k, float *a, float *b) {
   extern volatile __shared__ float sdata[];
   int     tid        = threadIdx.x,
           gridSize   = blockSize * gridDim.x,
           p          = blockIdx.x * blockSize + tid;
   float c;
   sdata[tid] = 0;
   while (p < n - k)
       {sdata[tid] += a[p]*b[p+k] + a[p+k]*b[p];
        p += gridSize;}
   __syncthreads();
   if (blockSize >= 512)
       { if (tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads(); }
   if (blockSize >= 256)
       { if (tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads(); }
   if (blockSize >= 128)
       { if (tid <  64) sdata[tid] += sdata[tid + 64]; __syncthreads(); }
   if (tid < 32)
       { if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
         if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
         if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
         if (blockSize >= 8)  sdata[tid] += sdata[tid + 4];
         if (blockSize >= 4)  sdata[tid] += sdata[tid + 2];
         if (blockSize >= 2)  sdata[tid] += sdata[tid + 1]; }
   if (tid == 0) {c[blockIdx.x] = 0.5f*sdata[0];}
   //if (!threadIdx.x) c = 0.5f*sdata[0];
   return c;
}

__global__ void parent_chebyprod(int n, float *a, float *b, float *c) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   float c_k;
   if (i < n) {
      c_k = child_sum <256> <<<(n + 256 - 1) / 256, 256, 256*sizeof(float)>>>
                (n, blockIdx.x * blockDim.x + threadIdx.x, a, b);
      float sum = 0.f;
      for (int k = 0; k <= i; k++) {sum += a[k]*b[i-k];}
      //hipDeviceSynchronize();
      //__syncthreads();
      c[i] = 0.5f*(sum + c_k);
   }
}

int main(void){
  clock_t tStart = clock();
  int N = 100;
  float *a, *b, *c, *d, *d_a, *d_b, *d_c, *d_d;
  a = (float*)malloc(N*sizeof(float));
  b = (float*)malloc(N*sizeof(float));
  c = (float*)malloc(N*sizeof(float));
  d = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_a, N*sizeof(float)); 
  hipMalloc(&d_b, N*sizeof(float));
  hipMalloc(&d_c, N*sizeof(float));
  hipMalloc(&d_d, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    a[i] = 0.1f;
    b[i] = 0.2f;
    d[i] = 0.0f;
  }
   
  for (int i = 0; i < N; i++) {
     float sum = 0.f;
     for (int k = 0; k < N-i; k++) {sum += a[k]*b[k+i]+a[k+i]*b[k];}
     for (int k = 0; k <= i; k++) {sum += a[k]*b[i-k];}
     d[i] = 0.5f*sum;
  }

  std::cout << "Vector d: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << d[k] << " ";
  std::cout <<"]\n";

  for (int i = 0; i < N; i++) {
     d[i] = 0.0f;
  }

  

  hipMemcpy(d_a, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(float), hipMemcpyHostToDevice);
  //hipMemcpy(d_d, d, N*sizeof(float), hipMemcpyHostToDevice);
  
  int blockSize, gridSize;
  // Number of threads in each thread block
  blockSize = 256;
  
  // Number of thread blocks in grid
  gridSize = (N + blockSize - 1) / blockSize;

  std::cout << "blockSize: " << blockSize << "\ngridSize: " << gridSize << "\n";
  
  // Perform chebyprod on N elements
  parent_chebyprod<<< gridSize, blockSize >>>(N, d_a, d_b, d_c);
  printf("Time taken: %.2fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
  
  hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);
  
  std::cout << "Vector c: [ ";
  for (int k = 0; k < 10; ++k)
    std::cout << c[k] << " ";
  std::cout <<"]\n";

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(a);
  free(b);
  free(c);
}
